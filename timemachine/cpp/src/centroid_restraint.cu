#include <chrono>
#include <iostream>
#include <vector>
#include <complex>
#include "centroid_restraint.hpp"
#include "gpu_utils.cuh"
#include "k_centroid_restraint.cuh"

namespace timemachine {

template <typename RealType>
CentroidRestraint<RealType>::CentroidRestraint(
    const std::vector<int> &group_a_idxs,
    const std::vector<int> &group_b_idxs,
    const double kb,
    const double b0
) : N_A_(group_a_idxs.size()),
    N_B_(group_b_idxs.size()),
    kb_(kb),
    b0_(b0) {

    gpuErrchk(hipMalloc(&d_group_a_idxs_, N_A_*sizeof(*d_group_a_idxs_)));
    gpuErrchk(hipMemcpy(d_group_a_idxs_, &group_a_idxs[0], N_A_*sizeof(*d_group_a_idxs_), hipMemcpyHostToDevice));

    gpuErrchk(hipMalloc(&d_group_b_idxs_, N_B_*sizeof(*d_group_b_idxs_)));
    gpuErrchk(hipMemcpy(d_group_b_idxs_, &group_b_idxs[0], N_B_*sizeof(*d_group_b_idxs_), hipMemcpyHostToDevice));

};

template <typename RealType>
CentroidRestraint<RealType>::~CentroidRestraint() {
    gpuErrchk(hipFree(d_group_a_idxs_));
    gpuErrchk(hipFree(d_group_b_idxs_));
};


template <typename RealType>
void CentroidRestraint<RealType>::execute_device(
        const int N,
        const int P,
        const double *d_x,
        const double *d_p,
        const double *d_box,
        const double lambda,
        unsigned long long *d_du_dx,
        double *d_du_dp,
        unsigned long long *d_du_dl,
        unsigned long long *d_u,
        hipStream_t stream) {

    int tpb = 32;

    k_centroid_restraint<RealType><<<1, tpb, 0, stream>>>(
        N,
        d_x,
        d_group_a_idxs_,
        d_group_b_idxs_,
        N_A_,
        N_B_,
        kb_,
        b0_,
        d_du_dx,
        d_u
    );
    gpuErrchk(hipPeekAtLastError());

};

template class CentroidRestraint<double>;
template class CentroidRestraint<float>;

} // namespace timemachine